#include "hip/hip_runtime.h"
#if defined(__HIPCC__) || defined(__CUDA_ARCH__) || defined(__CUDA_LIBDEVICE__)
#define __noinline__  __attribute__((noinline))
#endif /* __HIPCC__  || __CUDA_ARCH__ || __CUDA_LIBDEVICE__ */

#include "CUSPIS.cuh"
#include <iostream>

__global__ void axpy(float a, float* x, float* y) {
  y[threadIdx.x] = a * x[threadIdx.x];
}

int run_axpy(CUSPIS::cuspisRedundancyPolicy policy, float *host_x, int N) {
  volatile int blocks = 1;

  float a = 2.0f;
  float host_y[N];

  // Copy input data to device.
  float* device_x;
  float* device_y;
  CUSPIS::cuspisMalloc(&device_x, N * sizeof(float));
  CUSPIS::cuspisMalloc(&device_y, N * sizeof(float));
  CUSPIS::cuspisMemcpyToDevice(device_x, host_x, N * sizeof(float));

  // Launch the kernel.
  CUSPIS::Kernel<float, float*, float*> k(1, N, axpy, policy);
  k.launch(a, device_x, device_y);

  // Copy output data to host.
  hipDeviceSynchronize();
  CUSPIS::cuspisMemcpyToHost(host_y, device_y, N * sizeof(float));

  CUSPIS::cuspisFree(&device_x);
  CUSPIS::cuspisFree(&device_y);

  return 0;
}

int main(int argc, char* argv[]) {
    int SIZES = 1024;

    if (argc > 1) {
        SIZES = atoi(argv[2]);
    }

    float avg = 0.0;
    FILE *fp_blocks = fopen("axpy_b.txt", "w");
    FILE *fp_thread = fopen("axpy_t.txt", "w");
    FILE *fp_kernel = fopen("axpy_k.txt", "w");

    float *host_d = (float*) malloc(sizeof(float) * SIZES);

    for (int i=0; i<SIZES; i++) {
        host_d[i] = i;
    }

    // warm-up
    run_axpy(CUSPIS::cuspisRedundantBlocks, host_d, SIZES);

    for (int i=0; i<SIZES; i++) {
        float time;
        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        run_axpy(CUSPIS::cuspisRedundantBlocks, host_d, i);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);

        fprintf(fp_blocks, "%3.5f\n", time);
        avg = avg + (time - avg)/(i+1);
    }
    fclose(fp_blocks);
    printf("avg (redundant blocks):\t\t%3.5f\n", avg);

    avg = 0.0;
    hipDeviceReset();

    // warm-up
    run_axpy(CUSPIS::cuspisRedundantThreads, host_d, SIZES);

    for (int i=0; i<SIZES; i++) {
        float time;
        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        run_axpy(CUSPIS::cuspisRedundantThreads, host_d, i);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);

        fprintf(fp_thread, "%3.5f\n", time);
        avg = avg + (time - avg)/(i+1);
    }
    fclose(fp_thread);
    printf("avg (redundant threads):\t%3.5f\n", avg);

    avg = 0.0;
    hipDeviceReset();

    // warm-up
    run_axpy(CUSPIS::cuspisRedundantKernel, host_d, SIZES);

    for (int i=0; i<SIZES; i++) {
        float time;
        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        run_axpy(CUSPIS::cuspisRedundantKernel, host_d, i);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);

        fprintf(fp_kernel, "%3.5f\n", time);
        avg = avg + (time - avg)/(i+1);
    }
    fclose(fp_kernel);
    printf("avg (redundant kernels):\t%3.5f\n", avg);


}
