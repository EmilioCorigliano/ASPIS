#include "hip/hip_runtime.h"
/*
 Copyright (c) Loizos Koutsantonis <loizos.koutsantonis@uni.lu>

 Description : CUDA code implementing convolution of an image with a
 LoG kernel.
 Implemented for educational purposes.

 This program is free software: you can redistribute it and/or modify
 it under the terms of the NVIDIA Software License Agreement and CUDA
 Supplement to Software License Agreement.

 University of Luxembourg - HPC
 November 2020
*/

#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>

#include "CUSPIS.cuh"

#define pi 3.14159265359

/*
Function load_image:
Load BW Image from dat (Ascii) file (Host Function)
dim_x,dim_y: Image Dimensions
fname: fielename (char)
img: float vector containing the image pixels
*/
void load_image(char *fname, int dim_x, int dim_y, float *img) {
  FILE *fp;
  fp = fopen(fname, "r");

  int cnt = 0;
  for (int i = 0; i < dim_y; i++) {
    for (int j = 0; j < dim_x; j++) {
      fscanf(fp, "%f ", &img[i * dim_x + j]);
      cnt++;
    }

    while (fgetc(fp) != '\n') {
      continue;
    }
  }

  fclose(fp);
}

/*
Function save_image:
Save BW Image to dat (Ascii) file (Host Function)
dim_x,dim_y: Image Dimensions
fname: fielename (char)
img: float vector containing the image pixels
*/
void save_image(char *fname, int dim_x, int dim_y, float *img) {
  FILE *fp;

  fp = fopen(fname, "w");

  for (int i = 0; i < dim_y; i++) {
    for (int j = 0; j < dim_x; j++)
      fprintf(fp, "%10.3f ", img[i * dim_x + j]);
    fprintf(fp, "\n");
  }

  fclose(fp);
}

/*
Function calculate_kernel:
Calculate filter coefficients of LoG filter
and save them to a vector (Host Function)
kernel_size: Length of filter window in pixels (same for x and y)
sigma: sigma of the Gaussian kernel (float) given in pixels
kernel: float vector hosting the kernel coefficients
*/
void calculate_kernel(int kernel_size, float sigma, float *kernel) {

  int Nk2 = kernel_size * kernel_size;
  float x, y, center;

  center = (kernel_size - 1) / 2.0;

  for (int i = 0; i < Nk2; i++) {
    x = (float)(i % kernel_size) - center;
    y = (float)(i / kernel_size) - center;
    kernel[i] = -(1.0 / pi * pow(sigma, 4)) *
                (1.0 - 0.5 * (x * x + y * y) / (sigma * sigma)) *
                exp(-0.5 * (x * x + y * y) / (sigma * sigma));
  }
}

/*
Function conv_img_cpu:
Convolve image with the specified kernel  (Host Function)
img: float vector containing the original image pixels
kernel: float vector hosting the kernel coefficients
imgf: float vector containing the result of the convolution
dim_x,dim_y: Original Image Dimensions
kernel_size: Length of filter window in pixels (same for x and y)
*/
void conv_img_cpu(float *img, float *kernel, float *imgf, int dim_x, int dim_y,
                  int kernel_size) {

  float sum = 0;
  int center = (kernel_size - 1) / 2;
  ;
  int ii, jj;

  for (int i = 0; i < (dim_y - 0); i++){
    for (int j = 0; j < (dim_x - 0); j++) {
      sum = 0;
      for (int ki = 0; ki < kernel_size; ki++)
        for (int kj = 0; kj < kernel_size; kj++) {
          ii = kj + j - center;
          jj = ki + i - center;
          if ((i % dim_x) - center >= 0 && (i % dim_x) + center < dim_x &&
          (j % dim_y) - center >= 0 && (j % dim_y) + center < dim_y) {
            sum += img[jj * dim_x + ii] * kernel[ki * kernel_size + kj];
          }
          //sum += img[jj * dim_x + ii] * kernel[ki * kernel_size + kj];
        }
      imgf[i * dim_x + j] = sum;
    }
  }
}

/*
Function conv_img_cpu:
Convolve image with the specified kernel  (Device Function)
img: float vector containing the original image pixels
kernel: float vector hosting the kernel coefficients
imgf: float vector containing the result of the convolution
dim_x,dim_y: Original Image Dimensions
kernel_size: Length of filter window in pixels (same for x and y)
*/
__global__ void conv_img_gpu(float *img, float *kernel, float *imgf, int dim_x,
                             int dim_y, int kernel_size) {
  // each block is assigned to a row of an image, iy index of y value
  int iy = blockIdx.x + dim_y * (int)(threadIdx.x / dim_x);

  // each thread is assigned to a pixel of a row, ix index of x value
  int ix = threadIdx.x % dim_x;

  // idx global index (all blocks) of the image pixel
  int idx = iy * dim_x + ix;

  // center of kernel in both dimensions
  int center = (kernel_size - 1) / 2;

  // Auxiliary variables
  int ii, jj;
  float sum = 0.0;

  /*
  Convlution of image with the kernel
  Each thread computes the resulting pixel value
  from the convolution of the original image with the kernel;
  number of computations per thread = size_kernel^2
  The result is stored to imgf
  */

  for (int ki = 0; ki < kernel_size; ki++)
    for (int kj = 0; kj < kernel_size; kj++) {
      ii = kj + ix - center;
      jj = ki + iy - center;
      if ((ix % dim_x) - center >= 0 && (ix % dim_x) + center < dim_x &&
          (iy % dim_y) - center >= 0 && (iy % dim_y) + center < dim_y) {
        sum += img[jj * dim_x + ii] * kernel[ki * kernel_size + kj];
      }
    }

  imgf[idx] = sum;
}

void calculate_mean_std_dev(float arr[], int n) {
  float sum = 0.0;
  float variance = 0.0;
  float mean, std_dev;

  // Calculate the sum of elements
  for (int i = 0; i < n; i++) {
    sum += arr[i];
  }

  // Calculate the mean
  mean = sum / n;

  // Calculate the variance
  for (int i = 0; i < n; i++) {
    variance += pow(arr[i] - mean, 2);
  }

  // Calculate the standard deviation
  std_dev = sqrt(variance / n);

  printf("Mean: %f\nStd.Dev: %f\n", mean, std_dev);
}

void printDeviceProps() {
  hipError_t error = hipFree(0); // Initialize CUDA
  if (error != hipSuccess) {
    fprintf(stderr, "Could not initialize CUDA: %s\n", hipGetErrorName(error));
    return;
  }

  hipDeviceProp_t deviceProp;
  error = hipGetDeviceProperties(&deviceProp, 0); // Get properties of device 0
  if (error != hipSuccess) {
    fprintf(stderr, "Could not get device properties: %s\n",
            hipGetErrorName(error));
    return;
  }
  int maxGridDimX = deviceProp.maxGridSize[0];
  int maxGridDimY = deviceProp.maxGridSize[1];
  int maxGridDimZ = deviceProp.maxGridSize[2];
  int maxSurface1D = deviceProp.maxSurface1D;
  int maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
  int maxThreadsPerSM = deviceProp.maxBlocksPerMultiProcessor;
  int numSMs = deviceProp.multiProcessorCount;

  printf("Maximum grid dimensions: %d x %d x %d\n", maxGridDimX, maxGridDimY,
         maxGridDimZ);
  printf("Maximum surface 1D: %d\n", maxSurface1D);
  printf("Maximum threads per block: %d\n", maxThreadsPerBlock);
  printf("Maximum blocks per SM: %d\n", maxThreadsPerSM);
  printf("Number of SMs: %d\n", numSMs);
  printf("\n\n");
}

int main_gpu(int argc, char *argv[]) {
  float results[1000];
  CUSPIS::cuspisRedundancyPolicy policies[3] = {CUSPIS::cuspisRedundantThreads,
                                                CUSPIS::cuspisRedundantBlocks,
                                                CUSPIS::cuspisRedundantKernel};

  for (auto policy : policies) {

    std::string fname = "input.dat";

    for (int j = 0; j < 100; j++) {
      std::cout << "run " << j << "\r" << std::flush;
      hipEvent_t start, stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);
      float milliseconds = 0;
      int dim_x, dim_y;
      int kernel_size;
      float sigma;
      char finput[256], foutput[256];
      int Nblocks, Nthreads;

      sprintf(finput, fname.c_str());
      sprintf(foutput, "out.dat");

      dim_x = 64;
      dim_y = 64;

      kernel_size = 5;
      sigma = 0.8;

      /* Allocate CPU memory
          Vector Representation of Images and Kernel
          (Original Image, Kernel, Convoluted Image) */
      float *img, *imgf, *kernel;

      img = (float *)malloc(dim_x * dim_y * sizeof(float));
      imgf = (float *)malloc(dim_x * dim_y * sizeof(float));
      kernel = (float *)malloc(kernel_size * kernel_size * sizeof(float));

      /* Allocate GPU memory
          Vector Representation of Images and Kernel
          (Original Image, Kernel, Convoluted Image) */

      float *d_img, *d_imgf, *d_kernel;

      CUSPIS::cuspisMalloc(&d_img, dim_x * dim_y * sizeof(float));
      CUSPIS::cuspisMalloc(&d_imgf, dim_x * dim_y * sizeof(float));
      CUSPIS::cuspisMalloc(&d_kernel,
                            kernel_size * kernel_size * sizeof(float));

      load_image(finput, dim_x, dim_y, img);
      calculate_kernel(kernel_size, sigma, kernel);


      hipEventRecord(start);
      CUSPIS::cuspisMemcpyToDevice(d_img, img, dim_x * dim_y * sizeof(float));
      CUSPIS::cuspisMemcpyToDevice(d_kernel, kernel,
                                    kernel_size * kernel_size * sizeof(float));

      Nblocks = dim_y;
      Nthreads = dim_x;

      CUSPIS::Kernel<float *, float *, float *, int, int, int> k(
          Nblocks, Nthreads, conv_img_gpu, policy);

      // conv_img_cpu(img, kernel, imgf, dim_x, dim_y, kernel_size);
      for (int i = 0; i < 1; i++) {
        k.launch(d_img, d_kernel, d_imgf, dim_x, dim_y, kernel_size);
      }
      hipDeviceSynchronize();

      CUSPIS::cuspisMemcpyToHost(imgf, d_imgf, dim_x * dim_y * sizeof(float));

      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds, start, stop);

      save_image(foutput, dim_x, dim_y, imgf);

      free(img);
      free(imgf);
      free(kernel);

      CUSPIS::cuspisFree(&d_img);
      CUSPIS::cuspisFree(&d_imgf);
      CUSPIS::cuspisFree(&d_kernel);

      hipDeviceReset();

      results[j] = milliseconds;
    }
    std::cout << "Policy " << policy << "\n";
    std::cout << "Input file " << fname << "\n";
    calculate_mean_std_dev(results, 100);
    std::cout << "\n";
  }
  return 0;
}


int main_cpu(int argc, char *argv[]) {
  float results[1000];

  std::string fname = "input.dat";

  for (int j = 0; j < 100; j++) {
    std::cout << "run " << j << "\r" << std::flush;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    int dim_x, dim_y;
    int kernel_size;
    float sigma;
    char finput[256], foutput[256];

    sprintf(finput, fname.c_str());
    sprintf(foutput, "out.dat");

    dim_x = 64;
    dim_y = 64;

    kernel_size = 5;
    sigma = 0.8;

    /* Allocate CPU memory
        Vector Representation of Images and Kernel
        (Original Image, Kernel, Convoluted Image) */
    float *img, *imgf, *kernel;

    img = (float *)malloc(dim_x * dim_y * sizeof(float));
    imgf = (float *)malloc(dim_x * dim_y * sizeof(float));
    kernel = (float *)malloc(kernel_size * kernel_size * sizeof(float));

    load_image(finput, dim_x, dim_y, img);
    std::cout<< ("ciao\n");
    calculate_kernel(kernel_size, sigma, kernel);


    hipEventRecord(start);

    conv_img_cpu(img, kernel, imgf, dim_x, dim_y, kernel_size);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    save_image(foutput, dim_x, dim_y, imgf);

    free(img);
    free(imgf);
    free(kernel);

    results[j] = milliseconds;
  }
  std::cout << "Policy serial\n";
  std::cout << "Input file " << fname << "\n";
  calculate_mean_std_dev(results, 100);
  std::cout << "\n";
  
  return 0;
}

int main(int argc, char *argv[]) {
  if (argc != 2) {
    std::cout << "Please use two arguments (./out [cpu/gpu])\n";
    return 0;
  }
  if (strcmp(argv[1], "cpu") == 0)
    main_cpu(argc, argv);
  else if(strcmp(argv[1], "gpu") == 0)
    main_gpu(argc, argv);
  else 
    std::cout << "Please specify cpu or gpu\n";
}