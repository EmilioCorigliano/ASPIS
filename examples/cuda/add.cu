#include "hip/hip_runtime.h"
#include "CUSPIS.cuh"
#include <cstdio>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>

__global__ void add(int* a, int* b, int* c, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<N*CUSPIS::NUM_REPLICAS) {
    //if (i<N) {
        c[i] = a[i] + b[i];
    }
}

int perform_add(CUSPIS::cuspisRedundancyPolicy policy) {
    const int N = 1024;

    int *h_a, *h_b, *h_c;
    int *d_a, *d_b, *d_c;

    // host memory
    h_a = (int*) malloc(N * sizeof(int));
    h_b = (int*) malloc(N * sizeof(int));
    h_c = (int*) malloc(N * sizeof(int));

    // init data
    for (int i=0; i<N; i++) {
        h_a[i] = i;
        h_b[i] = i*2;
    }

    // device memory
    CUSPIS::cuspisMalloc(&d_a, N * sizeof(int));
    CUSPIS::cuspisMalloc(&d_b, N * sizeof(int));
    CUSPIS::cuspisMalloc(&d_c, N * sizeof(int));

    // copy to device
    CUSPIS::cuspisMemcpyToDevice(d_a, h_a, N * sizeof(int));
    CUSPIS::cuspisMemcpyToDevice(d_b, h_b, N * sizeof(int));

    // create and launch the kernel
    CUSPIS::Kernel<int*, int*, int*, int> k((N+255)/256, 256, add, policy);

    k.launch(d_a, d_b, d_c, N);

    hipDeviceSynchronize();

    // copy back to host
    CUSPIS::cuspisMemcpyToHost(h_c, d_c, N * sizeof(int));

    CUSPIS::cuspisFree(&d_a);
    CUSPIS::cuspisFree(&d_b);
    CUSPIS::cuspisFree(&d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}


int main(int argc, char* argv[]) {
    int NUM_ITER = 1024;

    if (argc > 1) {
        NUM_ITER = atoi(argv[1]);
    }

    float avg = 0.0;
    FILE *fp_blocks = fopen("add_b.txt", "w");
    FILE *fp_thread = fopen("add_t.txt", "w");
    FILE *fp_kernel = fopen("add_k.txt", "w");

    // warm-up
    perform_add(CUSPIS::cuspisRedundantBlocks);

    for (int i=0; i<NUM_ITER; i++) {
        float time;
        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        perform_add(CUSPIS::cuspisRedundantBlocks);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);

        fprintf(fp_blocks, "%3.5f\n", time);
        avg = avg + (time - avg)/(i+1);
    }
    fclose(fp_blocks);
    printf("avg (redundant blocks):\t\t%3.5f\n", avg);

    avg = 0.0;
    hipDeviceReset();

    // warm-up
    perform_add(CUSPIS::cuspisRedundantThreads);

    for (int i=0; i<NUM_ITER; i++) {
        float time;
        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        perform_add(CUSPIS::cuspisRedundantThreads);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);

        fprintf(fp_thread, "%3.5f\n", time);
        avg = avg + (time - avg)/(i+1);
    }
    fclose(fp_thread);
    printf("avg (redundant threads):\t%3.5f\n", avg);

    avg = 0.0;
    hipDeviceReset();

    // warm-up
    perform_add(CUSPIS::cuspisRedundantKernel);

    for (int i=0; i<NUM_ITER; i++) {
        float time;
        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        perform_add(CUSPIS::cuspisRedundantKernel);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);

        fprintf(fp_kernel, "%3.5f\n", time);
        avg = avg + (time - avg)/(i+1);
    }
    fclose(fp_kernel);
    printf("avg (redundant kernels):\t%3.5f\n", avg);


}